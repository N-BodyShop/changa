#include "hip/hip_runtime.h"
#ifdef CUDA_UNIT_TEST
# include "CUDAMoments.h"
#else
# include "cuda_typedef.h"
#endif

#ifdef GPU_LOCAL_TREE_WALK
#include "codes.h"
#endif //GPU_LOCAL_TREE_WALK

#ifdef CUDA_UNIT_TEST
__global__ void
#else
__device__ inline void __attribute__(( always_inline ))
#endif
CUDA_momEvalFmomrcm(const CudaMultipoleMoments* _m,
                    const CudaVector3D* _r,
                    cudatype dir,
                    CudaVector3D* out,
                    cudatype* pot)
{
  /* Added in these values for standalone function. */
  CudaMultipoleMoments m = *_m;
  CudaVector3D r = *_r;

  /* The following code is adapted from from `momEvalFmomrcm` in
     "moments.c"; the changes make the code work within the inputs
     available here, and (hopefullly) make the code a little more
     readable. */
  const cudatype onethird = 1.0 / 3.0;

  /* -> Build the reciprocal-of-radius and scaling-factor values. */
  cudatype
    /* in `momEvalFmomrcm`, `u` is a parameter, and the value passed a
       MultipoleMoments::radius instance (see point(s) of call at
       `nodeBucketForce` in "gravity.h").  `momEvalFmomrcm` also
       multiplies the parameter by `dir` prior to use. */
    u = dir * m.radius;

  /* -> Build the "g" terms, whose purpose is probably apparent to those
     who actually understand the math...  */
  cudatype
    g0 = dir,
    g2 = 3 * dir * u * u,
    g3 = 5 * g2 * u,
    g4 = 7 * g3 * u;


  /* -> "Calculate the trace-free distance terms." */
  cudatype
    x = r.x * dir,
    y = r.y * dir,
    z = r.z * dir,
    xx = 0.5f * x * x,
    xy = x * y,
    xz = x * z,
    yy = 0.5f * y * y,
    yz = y * z,
    zz = 0.5f * z * z,
    xxx = x * (onethird*xx - zz),
    xxz = z * (xx - onethird * zz),
    yyy = y * (onethird*yy - zz),
    yyz = z*(yy - onethird*zz);

  /* replace intermediates used above with their "final" values... */
  xx -= zz;
  yy -= zz;

  /* ...and finish with the trace-free terms. */
  cudatype
    xxy = y * xx,
    xyy = x * yy,
    xyz = xy * z;

  /* -> "Now calculate the interaction up to Hexadecapole order." */
  cudatype
    tx = g4 * ( m.xxxx*xxx + m.xyyy*yyy + m.xxxy*xxy +
                m.xxxz*xxz + m.xxyy*xyy + m.xxyz*xyz +
                m.xyyz*yyz ),
    ty = g4 * ( m.xyyy*xyy + m.xxxy*xxx + m.yyyy*yyy +
                m.yyyz*yyz + m.xxyy*xxy + m.xxyz*xxz +
                m.xyyz*xyz ),
    tz = g4 * (- m.xxxx*xxz - (m.xyyy + m.xxxy)*xyz
               - m.yyyy*yyz + m.xxxz*xxx + m.yyyz*yyy
               - m.xxyy*(xxz + yyz) + m.xxyz*xxy + m.xyyz*xyy);

  g4 = 0.25*(tx*x + ty*y + tz*z);

  /* Note that these variables have already been initialized; we're re-using them. */
  xxx = g3 * (m.xxx*xx + m.xyy*yy + m.xxy*xy + m.xxz*xz + m.xyz*yz);
  xxy = g3 * (m.xyy*xy + m.xxy*xx + m.yyy*yy + m.yyz*yz + m.xyz*xz);
  xxz = g3 * (-(m.xxx + m.xyy)*xz - (m.xxy + m.yyy)*yz + m.xxz*xx + m.yyz*yy + m.xyz*xy);

  g3 = onethird * (xxx*x + xxy*y + xxz*z);

  xx = g2*(m.xx*x + m.xy*y + m.xz*z);
  xy = g2*(m.yy*y + m.xy*x + m.yz*z);
  xz = g2*(-(m.xx + m.yy)*z + m.xz*x + m.yz*y);

  g2 = 0.5f*(xx*x + xy*y + xz*z);
  g0 *= m.totalMass;

  /* store the calculated potential  */
  *pot += -(g0 + g2 + g3 + g4);

  g0 += 5*g2 + 7*g3 + 9*g4;
  /* and the calculated acceleration. */
  out->x += dir*(xx + xxx + tx - x*g0);
  out->y += dir*(xy + xxy + ty - y*g0);
  out->z += dir*(xz + xxz + tz - z*g0);
}

#ifdef GPU_LOCAL_TREE_WALK

__device__ inline void __attribute__(( always_inline ))
addCudaVector3D(const CudaVector3D &a, const CudaVector3D &b, CudaVector3D &c) {
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  c.z = a.z + b.z;
}
__device__ inline void __attribute__(( always_inline ))
minusCudaVector3D(const CudaVector3D &a, const CudaVector3D &b, CudaVector3D &c) {
  c.x = a.x - b.x;
  c.y = a.y - b.y;
  c.z = a.z - b.z;
}
__device__ inline void __attribute__(( always_inline ))
assignCudaVector3D(const CudaVector3D &a, CudaVector3D &b) {
  b.x = a.x;
  b.y = a.y;
  b.z = a.z;
}

__device__ inline bool __attribute__(( always_inline ))
cuda_intersect(CUDABucketNode &b, CudaSphere &s) {
  cudatype dsq = 0.0;
  cudatype rsq = s.radius * s.radius;
  cudatype delta;
  if((delta = b.lesser_corner.x - s.origin.x) > 0)
      dsq += delta * delta;
    else if((delta = s.origin.x - b.greater_corner.x) > 0)
      dsq += delta * delta;
    if(rsq < dsq)
      return false;
    if((delta = b.lesser_corner.y - s.origin.y) > 0)
      dsq += delta * delta;
    else if((delta = s.origin.y - b.greater_corner.y) > 0)
      dsq += delta * delta;
    if(rsq < dsq)
      return false;
    if((delta = b.lesser_corner.z - s.origin.z) > 0)
      dsq += delta * delta;
    else if((delta = s.origin.z - b.greater_corner.z) > 0)
      dsq += delta * delta;
    return (dsq <= s.radius * s.radius);
}

__device__ inline bool __attribute__(( always_inline ))
CUDA_intersect(CudaSphere &s1, CudaSphere &s2) {
  CudaVector3D diff;
  cudatype dist;
  minusCudaVector3D(s1.origin, s2.origin, diff);
  dist = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
  return (dist <= (s1.radius + s2.radius) * (s1.radius + s2.radius));
}


__device__ inline bool __attribute__(( always_inline ))
CUDA_contains(const CudaSphere &s, const CudaVector3D &v) {
  CudaVector3D diff;
  cudatype dist;
  minusCudaVector3D(s.origin, v, diff);
  dist = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
  return (dist <= s.radius * s.radius);
}

__device__ inline int __attribute__(( always_inline ))
CUDA_openSoftening(CUDATreeNode &node, CUDABucketNode &myNode) {
  CudaSphere s;
  s.origin = node.cm;
  s.radius = 2.0 * node.soft;

  CudaSphere myS;
  myS.origin = myNode.cm;
  myS.radius = 2.0 * myNode.soft;

  if(CUDA_intersect(myS, s)) {
    return true;
  }
  return cuda_intersect(myNode, s);
}

__device__ inline int __attribute__(( always_inline ))
CUDA_openCriterionNode(CUDATreeNode &node,
                    CUDABucketNode &myNode,
                    int localIndex,
                    cudatype theta,
                    cudatype thetaMono) {
  const int nMinParticleNode = 6;
  const double openingGeometryFactor = 2.0 / sqrt(3.0);
  
  if(node.particleCount <= nMinParticleNode) {
    return 1;
  }

  // Note that some of this could be pre-calculated into an "opening radius"
  cudatype radius = openingGeometryFactor * node.radius / theta;

  if(radius < node.radius) {
    radius = node.radius;
  }

  CudaSphere s;
  s.origin = node.cm;
  s.radius = radius;

  if(cuda_intersect(myNode, s)) {
    return 1;
  } else {
#ifdef HEXADECAPOLE
    // Well separated, now check softening
    if(!CUDA_openSoftening(node, myNode)) {
      return 0;   // passed both tests: will be a Hex interaction
    } else {      // Open as monopole?
      radius = openingGeometryFactor * node.radius / thetaMono;
      CudaSphere sM;
      sM.origin = node.cm;
      sM.radius = radius;
      if(cuda_intersect(myNode, sM)) {
        return 1;
      }
      else {
        return 0;
      }
    }
#else
    return 0;
#endif //HEXADECAPOLE
  }
}

__device__ inline void __attribute__(( always_inline ))
CUDA_SPLINEQ(cudatype invr, cudatype r2, cudatype twoh, cudatype& a,
       cudatype& b,cudatype& c,cudatype& d) {
  cudatype u,dih,dir=(invr);
  if ((r2) < (twoh)*(twoh)) {
    dih = cudatype(2.0)/(twoh);
    u = dih/dir;
    if (u < cudatype(1.0)) {
      a = dih*(cudatype(7.0)/cudatype(5.0)
         - cudatype(2.0)/cudatype(3.0)*u*u
         + cudatype(3.0)/cudatype(10.0)*u*u*u*u
         - cudatype(1.0)/cudatype(10.0)*u*u*u*u*u);
      b = dih*dih*dih*(cudatype(4.0)/cudatype(3.0)
         - cudatype(6.0)/cudatype(5.0)*u*u
         + cudatype(1.0)/cudatype(2.0)*u*u*u);
      c = dih*dih*dih*dih*dih*(cudatype(12.0)/cudatype(5.0)
         - cudatype(3.0)/cudatype(2.0)*u);
      d = cudatype(3.0)/cudatype(2.0)*dih*dih*dih*dih*dih*dih*dir;
    } else {
      a = cudatype(-1.0)/cudatype(15.0)*dir
         + dih*(cudatype(8.0)/cudatype(5.0)
         - cudatype(4.0)/cudatype(3.0)*u*u + u*u*u
         - cudatype(3.0)/cudatype(10.0)*u*u*u*u
         + cudatype(1.0)/cudatype(30.0)*u*u*u*u*u);
      b = cudatype(-1.0)/cudatype(15.0)*dir*dir*dir
         + dih*dih*dih*(cudatype(8.0)/cudatype(3.0) - cudatype(3.0)*u
         + cudatype(6.0)/cudatype(5.0)*u*u
         - cudatype(1.0)/cudatype(6.0)*u*u*u);
      c = cudatype(-1.0)/cudatype(5.0)*dir*dir*dir*dir*dir
         + cudatype(3.0)*dih*dih*dih*dih*dir
         + dih*dih*dih*dih*dih*(cudatype(-12.0)/cudatype(5.0) 
         + cudatype(1.0)/cudatype(2.0)*u);
      d = -dir*dir*dir*dir*dir*dir*dir
         + cudatype(3.0)*dih*dih*dih*dih*dir*dir*dir
         - cudatype(1.0)/cudatype(2.0)*dih*dih*dih*dih*dih*dih*dir;
    }
  } else {
    a = dir;
    b = a*a*a;
    c = cudatype(3.0)*b*a*a;
    d = cudatype(5.0)*c*a*a;
  }
}

__device__ inline void __attribute__(( always_inline ))
CUDA_SPLINE(cudatype r2, cudatype twoh, cudatype &a, cudatype &b) {
  cudatype r, u,dih,dir;
  r = sqrt(r2);

  if (r < (twoh)) {
    dih = (2.0)/(twoh);
    u = r*dih;
    if (u < (1.0)) {
      a = dih*((7.0)/(5.0)
         - (2.0)/(3.0)*u*u
         + (3.0)/(10.0)*u*u*u*u
         - (1.0)/(10.0)*u*u*u*u*u);
      b = dih*dih*dih*((4.0)/(3.0)
         - (6.0)/(5.0)*u*u
         + (1.0)/(2.0)*u*u*u);
    } else {
      dir = (1.0)/r;
      a = (-1.0)/(15.0)*dir
         + dih*((8.0)/(5.0)
         - (4.0)/(3.0)*u*u + u*u*u
         - (3.0)/(10.0)*u*u*u*u
         + (1.0)/(30.0)*u*u*u*u*u);
      b = (-1.0)/(15.0)*dir*dir*dir
         + dih*dih*dih*((8.0)/(3.0) - (3.0)*u
         + (6.0)/(5.0)*u*u
         - (1.0)/(6.0)*u*u*u);
    }
  } else {
    a = (1.0)/r;
    b = a*a*a;
  }
}

// This function will to be simplified soon.
__device__ inline int __attribute__(( always_inline ))
CUDA_OptAction(int fakeOpen, int nodetype) {
  if (fakeOpen == 0) {
    if (nodetype == CudaInternal || nodetype == CudaBucket || nodetype == CudaBoundary || nodetype == CudaNonLocalBucket) {
      return COMPUTE;
    } else if (nodetype == CudaNonLocal || nodetype == CudaCached || nodetype == CudaCachedBucket || nodetype == CudaEmpty || nodetype == CudaCachedEmpty) {
      return DUMP;
    } else if (nodetype == CudaTop || nodetype == CudaInvalid) {
      return ERROR;
    } else {
      printf("ERROR in CUDA_OptAction\n");
      return -1;
    }
  } else {
    if (nodetype == CudaInternal || nodetype == CudaBoundary) {
      return KEEP;
    } else if (nodetype == CudaBucket) {
      return KEEP_LOCAL_BUCKET;
    } else if (nodetype == CudaNonLocal || nodetype == CudaNonLocalBucket || nodetype == CudaCachedBucket || nodetype == CudaCached || nodetype == CudaEmpty ||
              nodetype == CudaCachedEmpty) {
      return DUMP;
    } else if (nodetype == CudaTop || nodetype == CudaInvalid) {
      return ERROR;
    } else {
      printf("ERROR in CUDA_OptAction\n");
      return -1;
    }
  }
}

#endif //GPU_LOCAL_TREE_WALK


